#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file index_add.cu
 * \brief GPU implementation of index_add operator
 */

#include <hipcub/hipcub.hpp>
#include "./index_add-inl.h"
#include "../tensor/util/tensor_util-inl.cuh"
#include "../tensor/util/tensor_util-inl.h"

namespace mxnet {
namespace op {

template<typename DType, typename VType, int NDim>
struct IndexAddForwardGPUKernel {
  MSHADOW_XINLINE static void Map(size_t i, DType* out,
                                  const VType* val,
                                  const mshadow::Shape<NDim> a_tail_shape,
                                  const mshadow::Shape<NDim> a_pre_stride,
                                  const mshadow::Shape<NDim> val_stride,
                                  const mshadow::Shape<NDim> val_shape,
                                  const size_t a_tail_size, const int ind_num,
                                  const int ind_ndim, const int* ind_vec) {
    size_t id = 0;
    for (int dim = 0; dim < ind_ndim; ++dim) {
      id += a_pre_stride[dim] * ind_vec[dim * ind_num + i];
    }
    id *= a_tail_size;
    for (int _i = 0; _i < a_tail_size; ++_i) {
      mshadow::Shape<NDim> a_tail_id = mxnet_op::unravel(_i, a_tail_shape);
      mshadow::Shape<NDim> val_id;
      for (int _j = 0; _j < NDim; ++_j) {
        val_id[_j] = (val_shape[_j] == 1) ? 0 : a_tail_id[_j];
      }
      val_id[ind_ndim - 1] = (val_shape[ind_ndim - 1] == 1) ? 0 : i;
      size_t val_dest = mxnet_op::dot(val_id, val_stride);
      atomicAdd(&out[id + _i], static_cast<DType>(val[val_dest]));
    }
  }
};

template<typename xpu, typename DType, typename VType, int NDim>
void IndexAddForwardCalc(mshadow::Stream<xpu> *s,
                         const int ind_num, DType* out,
                         const VType* val,
                         const mshadow::Shape<NDim>& a_tail_shape,
                         const mshadow::Shape<NDim>& a_pre_stride,
                         const mshadow::Shape<NDim>& val_stride,
                         const mshadow::Shape<NDim>& val_shape,
                         const size_t a_tail_size,
                         const int ind_ndim, const int* ind_vec) {
  using namespace mxnet_op;
  using namespace mshadow;
  int* d_ind_vec;
  hipMalloc(reinterpret_cast<void**>(&d_ind_vec), sizeof(int) * ind_ndim * ind_num);
  hipMemcpy(d_ind_vec, ind_vec, sizeof(int) * ind_ndim * ind_num, hipMemcpyHostToDevice);
  Kernel<IndexAddForwardGPUKernel<DType, VType, NDim>, xpu>::Launch(
                                              s, ind_num, out, val,
                                              a_tail_shape, a_pre_stride,
                                              val_stride, val_shape,
                                              a_tail_size, ind_num,
                                              ind_ndim, d_ind_vec);
}


NNVM_REGISTER_OP(_npx_index_add)
.set_attr<FCompute>("FCompute<gpu>", IndexAddOpForward<gpu>);

}  // namespace op
}  // namespace mxnet

